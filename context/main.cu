#include <iostream>

#include <hip/hip_runtime.h>

#include "common.h"

int main(void)
{

    const size_t count = 1024 * 1024 * 128;

    DR_CHECK(hipInit(0));

    hipDevice_t dev;
    DR_CHECK(hipDeviceGet(&dev, 0));

    hipCtx_t c0, c1, c2;
    DR_CHECK(hipCtxCreate(&c0, 0, dev));
    DR_CHECK(hipCtxCreate(&c1, 0, dev));
    DR_CHECK(hipCtxCreate(&c2, 0, dev));

    float *p0, *p1, *p2;
    DR_CHECK(hipCtxSetCurrent(c0));
    RT_CHECK(hipMalloc(&p0, count));

    DR_CHECK(hipCtxSetCurrent(c1));
    RT_CHECK(hipMalloc(&p1, count));

    DR_CHECK(hipCtxSetCurrent(c2));
    RT_CHECK(hipMalloc(&p2, count));

    RT_CHECK(hipFree(p1));
    RT_CHECK(hipMalloc(&p1, count/2));

    std::cout << p0 << std::endl;
    std::cout << p1 << std::endl;
    std::cout << p2 << std::endl;

    std::cout << uintptr_t(p1) << " " << uintptr_t(p0) << " " << uintptr_t(p1) - uintptr_t(p0) << std::endl;
    std::cout << uintptr_t(p2) << " " << uintptr_t(p1) << " " << uintptr_t(p2) - uintptr_t(p1) << std::endl;
}